
/********************************************************************************************
* Implementing Graph Cuts on CUDA using algorithm given in CVGPU '08                       ** 
* paper "CUDA Cuts: Fast Graph Cuts on GPUs"                                               **  
*                                                                                          **   
* Copyright (c) 2008 International Institute of Information Technology.                    **  
* All rights reserved.                                                                     **  
*                                                                                          ** 
* Permission to use, copy, modify and distribute this software and its documentation for   ** 
* educational purpose is hereby granted without fee, provided that the above copyright     ** 
* notice and this permission notice appear in all copies of this software and that you do  **
* not sell the software.                                                                   **  
*                                                                                          **
* THE SOFTWARE IS PROVIDED "AS IS" AND WITHOUT WARRANTY OF ANY KIND,EXPRESS, IMPLIED OR    **
* OTHERWISE.                                                                               **  
*                                                                                          **
* Created By Vibhav Vineet.                                                                ** 
********************************************************************************************/

#include "CudaCuts.cu"
#include "Example.h"

#ifdef GEM5_FUSION
#include <stdint.h>
extern "C" {
void m5_work_begin(uint64_t workid, uint64_t threadid);
void m5_work_end(uint64_t workid, uint64_t threadid);
}
#endif

using namespace std; 

int main(int argc,char** argv)
{

	load_files(argv[1]) ;

#ifdef GEM5_FUSION
    m5_work_begin(0, 0);
#endif

	int initCheck = cudaCutsInit(width, height ,num_Labels) ;
	
	printf("Compute Capability %d\n",initCheck);

	if( initCheck > 0 )
	{
		printf("The grid is initialized successfully\n");
	}
	else 
		if( initCheck == -1 )
		{
			printf("Error: Please check the device present on the system\n");
		}
	

	int dataCheck   =  cudaCutsSetupDataTerm( dataTerm );

	if( dataCheck == 0 )
	{
		printf("The dataterm is set properly\n");
		
	}
	else 
		if( dataCheck == -1 )
		{
			printf("Error: Please check the device present on the system\n");
		}


	int smoothCheck =  cudaCutsSetupSmoothTerm( smoothTerm );


	if( smoothCheck == 0 )
	{
		printf("The smoothnessterm is set properly\n");
	}
	else
		if( smoothCheck == -1 )
		{
			printf("Error: Please check the device present on the system\n");
		}
	

	int hcueCheck   =  cudaCutsSetupHCue( hCue );

	if( hcueCheck == 0 )
	{
		printf("The HCue is set properly\n");
	}
	else
		if( hcueCheck == -1 )
		{
			printf("Error: Please check the device present on the system\n");
		}

	int vcueCheck   =  cudaCutsSetupVCue( vCue );


	if( vcueCheck == 0 )
	{
		printf("The VCue is set properly\n");
	}
	else 
		if( vcueCheck == -1 )
		{
			printf("Error: Please check the device present on the system\n");
		}


	int graphCheck = cudaCutsSetupGraph();

	if( graphCheck == 0 )
	{
		printf("The graph is constructed successfully\n");
	}
	else 
		if( graphCheck == -1 )
		{
			printf("Error: Please check the device present on the system\n");
		}

	int optimizeCheck = -1; 
	if( initCheck == 1 )
	{
		//CudaCuts involving atomic operations are called
		//optimizeCheck = cudaCutsAtomicOptimize();
		//CudaCuts involving stochastic operations are called
		optimizeCheck = cudaCutsStochasticOptimize();
	}


	if( optimizeCheck == 0 )
	{
		printf("The algorithm successfully converged\n");
	}
	else 
		if( optimizeCheck == -1 )
		{
			printf("Error: Please check the device present on the system\n");
		}

	int resultCheck = cudaCutsGetResult( );

	if( resultCheck == 0 )
	{
		printf("The pixel labels are successfully stored\n");
	}
	else 
		if( resultCheck == -1 )
		{
			printf("Error: Please check the device present on the system\n");
		}
		
	int energy = cudaCutsGetEnergy(); 


	initFinalImage();
	
	cudaCutsFreeMem();

#ifdef GEM5_FUSION
    m5_work_end(0, 0);
#endif
	
	exit(1);
	CUT_EXIT(argc,argv);
}


void load_files(char *filename)
{
	LoadDataFile(filename, width, height, num_Labels, dataTerm, smoothTerm, hCue, vCue);

}

void initFinalImage()
{
	out_pixel_values=(int**)malloc(sizeof(int*)*height);

	for(int i = 0 ; i < height ; i++ )
	{
		out_pixel_values[i] = (int*)malloc(sizeof(int) * width ) ;
		for(int j = 0 ; j < width ; j++ ) {
			out_pixel_values[i][j]=0;
		}
	}

	writeImage() ;
}

void writeImage()
{

	for(int i = 0 ; i <  graph_size1 ; i++)
	{

		int row = i / width1, col = i % width1 ;

		if(row >= 0 && col >= 0 && row <= height -1 && col <= width - 1 )
			out_pixel_values[row][col]=pixelLabel[i]*255;
	}

	write_image();
}

void write_image()
{

	FILE* fp=fopen("result_sponge/flower_cuda_test.pgm","w");

	fprintf(fp,"%c",'P');
	fprintf(fp,"%c",'2');
	fprintf(fp,"%c",'\n');
	fprintf(fp,"%d %c %d %c ",width,' ',height,'\n');
	fprintf(fp,"%d %c",255,'\n');

	for(int i=0;i<height;i++)
	{
		for(int j=0;j<width;j++)
		{
			fprintf(fp,"%d\n",out_pixel_values[i][j]);
		}
	}
	fclose(fp);
}



void LoadDataFile(char *filename, int &width, int &height, int &nLabels,
		int *&dataCostArray,
		int *&smoothCostArray,
		int *&hCue,
		int *&vCue)
{
	printf("enterd\n");
	
	FILE *fp = fopen(filename,"r");

	
	
	fscanf(fp,"%d %d %d",&width,&height,&nLabels);

	int i, n, x, y;
	int gt;
	for(i = 0; i < width * height; i++)
		fscanf(fp,"%d",&gt);

	dataCostArray = new int[width * height * nLabels];
	for(int c=0; c < nLabels; c++) {
		n = c;
		for(i = 0; i < width * height; i++) {
			fscanf(fp,"%d",&dataCostArray[n]);
			n += nLabels;
		}
	}

	hCue = new int[width * height];
	vCue = new int[width * height];

	n = 0;
	for(y = 0; y < height; y++) {
		for(x = 0; x < width-1; x++) {
			fscanf(fp,"%d",&hCue[n++]);
		}
		hCue[n++] = 0;
	}

	n = 0;
	for(y = 0; y < height-1; y++) {
		for(x = 0; x < width; x++) {
			fscanf(fp,"%d",&vCue[n++]);
		}
	}
	for(x = 0; x < width; x++) {
		vCue[n++] = 0;
	}

	fclose(fp);
	smoothCostArray = new int[nLabels * nLabels];

	smoothCostArray[0] = 0 ;
	smoothCostArray[1] = 1 ;
	smoothCostArray[2] = 1 ;
	smoothCostArray[3] = 0 ;


}



