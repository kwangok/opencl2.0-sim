/*****************************************************************************
 * A microbenchmark to test the performance of varying memory copy operations
 * including different sizes and different sources and destinations
 ****************************************************************************/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>
#include <string.h>
extern "C" {
    void m5_dumpreset_stats(uint64_t workid, uint64_t threadid);
}

int main(int argc, char** argv) {

    unsigned minimum_elements = 2;
    unsigned maximum_elements = 16 * 1024 * 1024;

    for (int index = 0; index < argc; ++index) {
        if (strcmp(argv[index], "-m") == 0) {
            if (argc > index+1) {
                minimum_elements = atoi(argv[++index]);
            } else {
                printf("ERROR: Must specify minimum array size to -m option\n");
                exit(0);
            }
        } else if (strcmp(argv[index], "-M") == 0) {
            if (argc > index+1) {
                maximum_elements = atoi(argv[++index]);
            } else {
                printf("ERROR: Must specify maximum array size to -M option\n");
                exit(0);
            }
        }
    }

    bool power_two = true;
    unsigned *array = (unsigned*) malloc(maximum_elements * sizeof(unsigned));
    unsigned *d_array;
    hipMalloc(&d_array, maximum_elements * sizeof(unsigned));

    printf("Testing copy host-to-device:\nSize:\n");
    m5_dumpreset_stats(0, 0);
    for (unsigned i = minimum_elements; i <= maximum_elements;) {
        printf("%d\n", i);
        hipMemcpy(d_array, array, i * sizeof(unsigned), hipMemcpyHostToDevice);
        m5_dumpreset_stats(0, 0);
        if (power_two) {
            if (i > 1) {
                i = 3 * i / 2;
                power_two = false;
            } else {
                i = 2;
            }
        } else {
            i = 4 * i / 3;
            power_two = true;
        }
    }

    printf("\n\nTesting copy device-to-host:\nSize:\n");
    m5_dumpreset_stats(0, 0);
    for (unsigned i = minimum_elements; i <= maximum_elements;) {
        printf("%d\n", i);
        hipMemcpy(array, d_array, i * sizeof(unsigned), hipMemcpyDeviceToHost);
        m5_dumpreset_stats(0, 0);
        if (power_two) {
            if (i > 1) {
                i = 3 * i / 2;
                power_two = false;
            } else {
                i = 2;
            }
        } else {
            i = 4 * i / 3;
            power_two = true;
        }
    }
    printf("\n");
    free(array);
    hipFree(d_array);

    return 0;
}
