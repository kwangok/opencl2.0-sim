/***************************************************************************
 * cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ***************************************************************************/

/* 
 * Main entry of dense matrix-matrix multiplication kernel
 */

#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>
#include <malloc.h>
#include <vector>
#include <assert.h>

#include "../common/parboil.h"
#include <iostream>
#include "sgemm_kernel.cu"

// I/O routines
extern bool readColMajorMatrixFile(const char *fn, int &nr_row, int &nr_col, std::vector<float>&v);
extern bool writeColMajorMatrixFile(const char *fn, int, int, std::vector<float>&);

extern "C"
void computeGold(float *, const float*, const float*, unsigned int, unsigned int, unsigned int);

int
main (int argc, char *argv[]) {
	struct pb_Parameters *params = (struct pb_Parameters *)malloc(sizeof(struct pb_Parameters));;
	struct pb_TimerSet timers;
	
	float *dA, *dB, *dC;
	size_t A_sz, B_sz, C_sz;
	int matArow, matAcol;
	int matBrow, matBcol;
	std::vector<float> matA, matBT;
	
	pb_InitializeTimerSet(&timers);
	
	/* Read command line. Expect 3 inputs: A, B and B^T 
	 *     in column-major layout*/

	params->outFile = NULL;

	if ( argc != 4 ) {
		fprintf(stderr, "Expecting three input filenames got:\n");
		exit(-1);
	}
	fprintf(stderr, "In-File 1: %s\n", argv[1]);
	fprintf(stderr, "In-File 2: %s\n", argv[2]);
	fprintf(stderr, "In-File 3: %s\n", argv[3]);

	/* Read in data */
	pb_SwitchToTimer(&timers, pb_TimerID_IO);
	
	// load A
	readColMajorMatrixFile(argv[1], matArow, matAcol, matA);
	// copy A to device memory
	A_sz = matArow*matAcol*sizeof(float);

	// load B^T
	readColMajorMatrixFile(argv[3], matBcol, matBrow, matBT);

	pb_SwitchToTimer( &timers, pb_TimerID_COMPUTE );
	B_sz = matBrow*matBcol*sizeof(float);
	
	// allocate space for C
	C_sz = matArow*matBcol*sizeof(float);
	
	// CUDA memory allocation
	std::vector<float> matC(matArow*matBcol);
//	hipMalloc((void**)&dA, A_sz);
//	hipMalloc((void**)&dB, B_sz);
//	hipMalloc((void**)&dC, C_sz);

	dA = (float *)malloc(A_sz);
	dB = (float *)malloc(B_sz);
	dC = (float *)malloc(C_sz);

	assert(dA && dB && dC);

	// Copy A and B^T into device memory
	pb_SwitchToTimer( &timers, pb_TimerID_COPY );

//	hipMemcpy(dA, &matA.front(), A_sz, hipMemcpyHostToDevice); 
//	hipMemcpy(dB, &matBT.front(), B_sz, hipMemcpyHostToDevice); 

	memcpy(dA, &matA.front(), A_sz); 
	memcpy(dB, &matBT.front(), B_sz); 

	pb_SwitchToTimer( &timers, pb_TimerID_GPU );
	
	// Use standard sgemm interface
	regtileSgemm('N', 'T', matArow, matBcol, matAcol, 1.0f, \
	dA, matArow, dB, matBcol, 0.0f, dC, matArow);
	
	if (params->outFile) {
		pb_SwitchToTimer( &timers, pb_TimerID_COPY );
//		hipMemcpy(&matC.front(), dC, C_sz, hipMemcpyDeviceToHost);
		memcpy(&matC.front(), dC, C_sz);
		/* Write C to file */
		pb_SwitchToTimer(&timers, pb_TimerID_IO);
		writeColMajorMatrixFile(params->outFile,matArow, matBcol, matC); 
	}
	
	pb_SwitchToTimer(&timers, pb_TimerID_NONE);
	
	double GPUtime = pb_GetElapsedTime(&(timers.timers[pb_TimerID_GPU]));
	std::cout<< "GFLOPs = " << 2.* matArow * matBcol * matAcol/GPUtime/1e9 << std::endl;
	pb_PrintTimerSet(&timers);
	pb_FreeParameters(params);
//	hipFree(dA);
//	hipFree(dB);
//	hipFree(dC);

	free(dA);
	free(dB);
	free(dC);

	return 0;
}
